#include <hipblas.h>

#include "mc_bench_cublas.h"

int get_state_cublas(
    double *initial_state,
    double *delta,
    double *transition,
    uint64_t n_state,
    uint64_t n_iter,
    double *out) {
    double *d_out_odd = 0;
    hipblasHandle_t handle;
    hipMalloc(&d_out_odd, n_state * sizeof(double));
    hipMemcpy(
        d_out_odd,
        initial_state,
        n_state * sizeof(double),
        hipMemcpyHostToDevice
    );
    double *d_delta = 0;
    hipMalloc(&d_delta, n_state * sizeof(double));
    hipMemcpy(
        d_delta,
        delta,
        n_state * sizeof(double),
        hipMemcpyHostToDevice
    );
    double *d_transition = 0;
    hipMalloc(&d_transition, n_state * n_state * sizeof(double));
    hipMemcpy(
        d_transition,
        transition,
        n_state * n_state * sizeof(double),
        hipMemcpyHostToDevice
    );
    double *d_out_even = 0;
    hipMalloc(&d_out_even, n_state * sizeof(double));
    double alpha = 1;
    double beta = 0;
    hipblasCreate(&handle);
    for (uint64_t i = 0; i < n_iter; ++i) {
        hipblasDaxpy(
            handle,
            n_state,
            &alpha,
            d_delta,
            1,
            i & 1 ? d_out_even : d_out_odd,
            1
        );
        hipblasDgemm(
            handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            n_state,
            1,
            n_state,
            &alpha,
            d_transition,
            n_state,
            i & 1 ? d_out_even : d_out_odd,
            n_state,
            &beta,
            i & 1 ? d_out_odd : d_out_even,
            n_state
        );
    }
    hipblasDestroy(handle);
    hipFree(d_delta);
    hipFree(d_transition);
    hipMemcpy(
        out,
        n_iter & 1 ? d_out_even : d_out_odd,
        n_state * sizeof(double),
        hipMemcpyDeviceToHost
    );
    hipFree(d_out_odd);
    hipFree(d_out_even);
    return 0;
}
